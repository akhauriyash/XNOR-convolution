#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hipblas.h>

using namespace std;


__global__ void bin(float* Img_d, unsigned int *Ker_d, float* Out_d, float* coeffD_img,
					 float* coeffD_ker, int w_img, int h_img, int w_ker,
					int h_ker, int inp_ch, int ker_depth, int dtypelen){

	__shared__ unsigned int x; x = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ unsigned int y; y = blockIdx.y * blockDim.y + threadIdx.y;
	__shared__ unsigned int pos; pos = blockIdx.x + blockIdx.y*16;
	__shared__ unsigned int idx;
	__shared__ unsigned int accum[256];
	
	idx = x + y*(w_img - w_ker + 1);

	if((x < (w_img - w_ker + 1) && (y < (h_img - h_ker + 1)))){
		for(int kd = 0; kd < ker_depth; kd++){
			for(int ic = 0; ic < inp_ch; ic++){
				accum[pos] = 0;
				for(int dx = 0; dx < w_ker; dx++){
					for(int dy = 0; dy < h_ker; dy++){
						accum[pos] = accum[pos] | ((Img_d[ic*w_img*h_img + x + dx + w_img*(y+dy)] >= 0) << ((w_ker*h_ker - 1) - (dx + w_ker*dy)));
					}
				}
				Out_d[idx + kd*(w_img-w_ker+1)*(h_img-h_ker+1)] += coeffD_ker[ic+kd*inp_ch]*coeffD_img[ic]*(2*(dtypelen - (__popc(accum[pos]^Ker_d[ic+kd*inp_ch]))) - (dtypelen - w_ker*h_ker));
			}
		}
	}
	__syncthreads();
}	

int main()	{
	hipError_t a, b, c, d, e, f;
	
	//		Input dimensions

	int w_img = 256;	int h_img = 256;	int inp_ch = 256;
	int w_ker = 4;		int h_ker = 4;		int ker_depth = 256;

	printf("\n\nw_img %d h_img %d inp_ch %d w_ker %d h_ker %d ker_depth %d\n\n", w_img, h_img, inp_ch, w_ker, h_ker, ker_depth);

	//		HOST Memory allocation
	
	float* Img_h = (float *) malloc(w_img*h_img*inp_ch*sizeof(float));
	float* Ker_h = (float *) malloc(w_ker*h_ker*inp_ch*ker_depth*sizeof(float));
	float* Out_h = (float *) malloc((w_img - w_ker + 1)*(h_img - h_ker + 1)*ker_depth*sizeof(float));
	float* Out_h2 = (float *) malloc((w_img - w_ker + 1)*(h_img - h_ker + 1)*ker_depth*sizeof(float));
	float* coeffH_ker = (float *) malloc(inp_ch*ker_depth*sizeof(float));
	float* coeffH_img = (float *) malloc(inp_ch*sizeof(float));
 	for (int i = 0; i < w_img*h_img*inp_ch ; i ++) {
		double x = (double)rand() / RAND_MAX;
		Img_h[i] = (x > 0.5) ? 1 : -1;			
	}
	for (int i = 0; i < w_ker*h_ker*inp_ch*ker_depth ; i ++) {
		double x = (double)rand() / RAND_MAX;
		Ker_h[i] = (x < 0.5) ? -1 : 1;
	}
	for(int i = 0; i < inp_ch; i++){
		double x = (double)rand() / RAND_MAX;
		coeffH_img[i] = x;
		for(int j = 0; j < ker_depth; j++){
			double x = (double)rand() / RAND_MAX;
			coeffH_ker[i*j] = x;
		}
	}

	//		DEVICE Memory allocation 

	float *Img_d, *Out_d;	unsigned int *Ker_d;		float *coeffD_ker, *coeffD_img;

	float *Kers;
	hipMalloc(&Kers, w_ker*h_ker*inp_ch*ker_depth*sizeof(float));
	hipMemcpy(Kers, Ker_h, w_ker*h_ker*inp_ch*ker_depth*sizeof(float), hipMemcpyHostToDevice);

	a = hipMalloc(&Img_d, w_img*h_img*inp_ch*sizeof(float));												//	FP Image memory block DEVICE
	b = hipMalloc(&Ker_d, inp_ch*ker_depth*sizeof(unsigned int));											//	Ker_d cudaMemcpy from Kconc
	c = hipMalloc(&Out_d, (w_img - w_ker + 1)*(h_img - h_ker + 1)*ker_depth*sizeof(float));				//	FP Output memory block DEVICE
	d = hipMemcpy(Img_d, Img_h, w_img*h_img*inp_ch*sizeof(float), hipMemcpyHostToDevice);					//	FP Image copy from HOST to DEVICE
	e = hipMemset(Ker_d, 0, inp_ch*ker_depth*sizeof(unsigned int));										//	Memset (temporary)
	f = hipMemset(Out_d, 0, (w_img - w_ker + 1)*(h_img - h_ker + 1)*ker_depth*sizeof(float));				//	Memset (temporary)
	cout << "Mallocs Memcpy & Memset:\t "<< a << b << c << d << e << f << "\n";
	unsigned int *Img_conc;																					//
	a = hipMalloc(&Img_conc, (w_img - w_ker + 1)*(h_img - h_ker + 1)*ker_depth*sizeof(unsigned int));		//	Uint Image memory block DEVICE
	b = hipMemset(Img_conc, 0, (w_img - w_ker + 1)*(h_img - h_ker + 1)*ker_depth*sizeof(unsigned int));	//	Uint Image memory setting
	c = hipMalloc(&coeffD_img, inp_ch*sizeof(float));
	d = hipMalloc(&coeffD_ker, inp_ch*ker_depth*sizeof(float));
	e = hipMemcpy(coeffD_img, coeffH_img, inp_ch*sizeof(float), hipMemcpyHostToDevice);
	f = hipMemcpy(coeffD_ker, coeffH_ker, inp_ch*ker_depth*sizeof(float), hipMemcpyHostToDevice);

	unsigned int* Ker_conc = (unsigned int *) malloc(inp_ch*ker_depth*sizeof(unsigned int));				//	Memset (IMPORTANT)
	if(Ker_conc == NULL){		printf("Ker_conc MALLOC FAILURE\n");	} 
			else 		{		printf("Ker_conc MALLOC SUCCESS\n");	}
	cout << "Img_conc malloc stat:\t" << a << b << "\n" << "Ker&Img malloc & memcpy stat:\t" << c << d 
						<< e << f << "\n";

	int blockx = 16;				int blocky = 16;														//	Block config (Is maximum)
	dim3 block(blockx, blocky);		dim3 grid(w_img/blockx + 1, h_img/blocky + 1);							//	Grid config

	auto conv_xnor = [&](){
		//		Concatenate kernels to unsigned int array Kconc[inp_ch*ker_depth]
		//		cudaMalloc contiguous memory block for array
		//		cudaMemcpy Kconc to Ker_d
		// unsigned int Kconc[ker_depth*inp_ch] = {0};
		unsigned int * Kconc = (unsigned int *) malloc(ker_depth*inp_ch*sizeof(unsigned int));
		if(Kconc == NULL){
			printf("Kconc MALLOC FAILURE\n");
		} else {printf("Kconc MALLOC SUCCESS\n");}

		//		Data arrangement				(w_ker, h_ker, inp_ch, ker_depth)
		for(int kd = 0; kd < ker_depth; kd++){
			for(int ic = 0; ic < inp_ch; ic++){
				for(int shift = 0; shift < w_ker*h_ker; shift++){
					Kconc[ic + inp_ch*kd] = Kconc[ic + inp_ch*kd] | ((Ker_h[shift + ic*(w_ker*h_ker) + kd*(ic*w_ker*h_ker)]>0) << (w_ker*h_ker - 1 - shift));
				}
				for(int shift = w_ker*h_ker; shift < 32; shift++){											//	Handle 32 to sizeof(dtype) in *bits*
					Kconc[ic + inp_ch*kd] = (Kconc[ic + inp_ch*kd] | (1<<shift));
				}
			}
		} 
		// for(int i = 0; i < ker_depth*inp_ch; i++){		cout << Kconc[i] << "\tCheck\n";	}
		a = hipMemcpy(Ker_d, Kconc, inp_ch*ker_depth*sizeof(unsigned int), hipMemcpyHostToDevice);
		cout << "Host to device Kconc-Ker_d:\t" << a << "\n";

		//		CudaMalloc and cudaMemcpy for Kernels on DEVICE is DONE Img_d is ALLOCATED
		//		call binConv function
		//		cudaDeviceSynchronize, assign result contiguous memory block on HOST
		//		cudaMemcpy result 

		int dtypelen = 32;																				//	Handle dtypelen to wk*hk bounding 
		hipDeviceSynchronize();

		bin<<<grid, block>>>(Img_d, Ker_d, Out_d, coeffD_img, coeffD_ker, w_img, h_img, w_ker, h_ker, inp_ch, ker_depth, dtypelen);

		hipDeviceSynchronize();

		a = hipMemcpy(Out_h, Out_d, (w_img - w_ker + 1)*(h_img - h_ker + 1)*ker_depth*sizeof(float), hipMemcpyDeviceToHost);

		cout << "bin Memcpy result:\t" << a << "\n";

		for(int i = 0; i < 25; i++){cout << Out_h[i] << " ";}		printf("\n");
	};
	
	conv_xnor();
}
